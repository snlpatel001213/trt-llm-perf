#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/speculativeDecoding/common.h"
#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm::kernels::speculative_decoding
{
template <int32_t BLOCK_SIZE>
__global__ void packAcceptedPaths(SizeType32* acceptedLengthsCumSum, SizeType32* pathsOffsets,
    SizeType32 const* acceptedLengths, SizeType32 const* bestPathIds, SizeType32 const* paths,
    SizeType32 const* batchSlots, SizeType32 batchSize, SizeType32 numPaths, SizeType32 maxPathLen,
    bool isPathsLinearBatchIdx)
{
    // Specialize BlockScan for a 1D block of 128 threads of type int
    typedef hipcub::BlockScan<SizeType32, BLOCK_SIZE> BlockScan;

    // Allocate shared memory for BlockScan
    __shared__ typename BlockScan::TempStorage tempStorage;
    auto const batchSizeRounded = ((batchSize + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;
    __shared__ SizeType32 currentCumSum;
    if (threadIdx.x == 0)
    {
        currentCumSum = 0;
    }

    __syncthreads();

    for (auto bi = static_cast<SizeType32>(threadIdx.x); bi < batchSizeRounded;
         bi += static_cast<SizeType32>(blockDim.x))
    {
        auto const valid = bi < batchSize;
        auto const batchSlot = valid ? batchSlots[bi] : 0;
        auto const acceptedLen = valid ? acceptedLengths[batchSlot] - 1 : 0;
        SizeType32 cumSum;
        BlockScan(tempStorage).ExclusiveSum(acceptedLen + currentCumSum, cumSum);
        if (threadIdx.x == blockDim.x - 1)
        {
            currentCumSum = cumSum;
        }
        __syncthreads();

        if (valid)
        {
            acceptedLengthsCumSum[bi] = cumSum;
            auto const pathBatchIdx = isPathsLinearBatchIdx ? bi : batchSlot;
            auto const bestPathIdx = bestPathIds[pathBatchIdx];
            auto const pathIdx = flat_index3(pathBatchIdx, bestPathIdx, 0, numPaths, maxPathLen);
            for (SizeType32 ti = 0; ti < acceptedLen; ++ti)
            {
                pathsOffsets[cumSum + ti] = paths[pathIdx + ti + 1] - 1;
            }
        }
    }
    if (threadIdx.x == 0)
    {
        acceptedLengthsCumSum[batchSize] = currentCumSum;
    }
}

void invokePackAcceptedPaths(SizeType32* acceptedLengthsCumSum, SizeType32* pathsOffsets,
    SizeType32 const* acceptedLengths, SizeType32 const* bestPathIds, SizeType32 const* paths,
    SizeType32 const* batchSlots, SizeType32 batchSize, SizeType32 numPaths, SizeType32 maxPathLen,
    bool isPathsLinearBatchIdx, hipStream_t stream)
{
    constexpr SizeType32 BLOCK_SIZE = 1024;
    packAcceptedPaths<BLOCK_SIZE><<<1, BLOCK_SIZE, 0, stream>>>(acceptedLengthsCumSum, pathsOffsets, acceptedLengths,
        bestPathIds, paths, batchSlots, batchSize, numPaths, maxPathLen, isPathsLinearBatchIdx);
}

namespace
{
__device__ __forceinline__ int4 reduceMaxInt4(int4 const& a, int4 const& b)
{
    return a.x >= b.x ? a : b;
}

template <typename T, SizeType32 BLOCK_SIZE>
__global__ void acceptDraftTokensByIdsWithPaths(TokenIdType* outputIds, TokenIdType const* draftIds,
    TokenIdType const* targetIds, SizeType32* sequenceLengths, SizeType32* acceptedLengths,
    FinishedState* finishedFinal, SizeType32 const* batchSlots, SizeType32 const* paths, TokenIdType const* endIds,
    T const** medusaLogits, T const** logitsPtrs, SizeType32* curTokensPerStep, SizeType32 const* targetTokensPerStep,
    SizeType32* bestPathIds, SizeType32 batchSize, SizeType32 vocabSize, SizeType32 maxBatchSize, SizeType32 maxSeqLen,
    SizeType32 maxDraftPathLen, SizeType32 maxDecodingTokens)
{
    auto const batchIdx = static_cast<SizeType32>(blockIdx.x);
    auto const batchSlot = batchSlots == nullptr ? batchIdx : batchSlots[batchIdx];
    auto const inputLength = sequenceLengths == nullptr ? 0 : sequenceLengths[batchSlot];
    auto const endId = endIds == nullptr ? -1 : endIds[batchSlot];
    auto const numTokensPerStep = curTokensPerStep == nullptr ? maxDecodingTokens : curTokensPerStep[batchSlot];
    auto const maxPathLen = maxDraftPathLen + 1;

    int4 partialMax{-1, -1, 0, 0};
    // Go over different paths and construct implicit sequences
    for (auto pathIdx = static_cast<SizeType32>(threadIdx.x); pathIdx < maxDecodingTokens;
         pathIdx += static_cast<SizeType32>(blockDim.x))
    {
        auto acceptedLength = maxPathLen;
        auto const pathOffset = flat_index3(batchSlot, pathIdx, 0, maxDecodingTokens, maxPathLen);
        bool hasEnd = false;

        auto const tokenId = paths[pathOffset];
        // Continue if path does not exist
        if (tokenId == -1)
        {
            continue;
        }
        auto const targetTokenIdx = batchSlot * maxDecodingTokens + tokenId;
        auto targetToken = targetIds[targetTokenIdx];
        auto nextIdx = tokenId;

        // Go along the path
        for (SizeType32 ti = 1; ti < maxPathLen; ++ti)
        {
            auto const tokenId = paths[pathOffset + ti];
            // Break if path terminates
            if (tokenId == -1)
            {
                hasEnd = endIds == nullptr ? false
                                           : targetToken == endId; // check if last token is EOS when path terminates.
                acceptedLength = hasEnd ? ti - 1 : ti;
                break;
            }
            auto const targetTokenIdx = batchSlot * maxDecodingTokens + tokenId;
            auto const draftTokenIdx = batchSlot * (maxDecodingTokens - 1) + tokenId - 1;
            // In context phase, no draft tokens are given. Set draft token to -1 to get guaranteed rejection
            auto const draftToken = tokenId >= numTokensPerStep ? -1 : draftIds[draftTokenIdx];
            // Check if draft tokens are the same as target tokens
            bool const accepted = draftToken == targetToken;
            hasEnd = endIds == nullptr ? false : targetToken == endId;
            if (!accepted || hasEnd)
            {
                acceptedLength = hasEnd ? ti - 1 : ti;
                break;
            }
            targetToken = targetIds[targetTokenIdx];
            nextIdx = tokenId;
        }
        // Get longest path of the thread
        if (partialMax.x < acceptedLength)
        {
            partialMax.x = acceptedLength;
            partialMax.y = pathIdx;
            partialMax.z = hasEnd;
            partialMax.w = nextIdx;
        }
    }

    // Get the longest path of the block (request)
    typedef hipcub::BlockReduce<int4, BLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tempStorage;
    int4 total = BlockReduce(tempStorage).Reduce(partialMax, reduceMaxInt4);

    __shared__ int4 totalShared;
    if (threadIdx.x == 0)
    {
        totalShared = total;
    }

    __syncthreads();

    auto const acceptedLength = totalShared.x;
    auto const bestPathIdx = totalShared.y;
    auto const bestNextIdx = numTokensPerStep == 1 ? 0 : totalShared.w;
    auto const pathOffset = flat_index3(batchSlot, bestPathIdx, 0, maxDecodingTokens, maxPathLen);
    for (auto ti = static_cast<SizeType32>(threadIdx.x); ti < acceptedLength; ti += static_cast<SizeType32>(blockDim.x))
    {
        auto const tokenId = paths[pathOffset + ti];
        auto const targetSrcTokenIdx = batchSlot * maxDecodingTokens + tokenId;
        auto const outputTokenIdx = batchSlot * maxSeqLen + inputLength + ti;
        auto const targetToken = targetIds[targetSrcTokenIdx];
        // Copy accepted tokens to the sequence with draft tokens (outputIds === outputIds)
        outputIds[outputTokenIdx] = targetToken;
    }

    // Leading thread reconstructs winning path and sets new data
    if (threadIdx.x == 0)
    {
        auto const hasEnd = totalShared.z;
        // Set end condition
        if (hasEnd && finishedFinal)
        {
            finishedFinal[batchSlot].setFinishedEOS();
        }
        // Make correction to the sequence length
        if (sequenceLengths)
        {
            sequenceLengths[batchSlot] += acceptedLength;
        }
        acceptedLengths[batchSlot] = acceptedLength;
        // In Medusa decoding step, number of draft tokens is 0 and must be updated for the next steps
        if (curTokensPerStep && targetTokensPerStep && numTokensPerStep == 1)
        {
            curTokensPerStep[batchSlot] = targetTokensPerStep[batchSlot];
        }
        bestPathIds[batchSlot] = bestPathIdx;
    }

    // Prepare logits pointers to respective logits from Medusa Heads for the all-top-K sampling kernel
    if (medusaLogits && logitsPtrs)
    {
        for (auto hi = static_cast<SizeType32>(threadIdx.x); hi < maxDraftPathLen;
             hi += static_cast<SizeType32>(blockDim.x))
        {
            logitsPtrs[batchIdx * maxDraftPathLen + hi]
                = medusaLogits[batchSlot * maxDraftPathLen + hi] + flat_index2(bestNextIdx, 0, vocabSize);
        }
    }
}
} // namespace

template <typename T>
void acceptDraftTokensByIdsWithPaths(AcceptDraftTokensByIdsWithPathsParams<T> const& params)
{
    constexpr SizeType32 BLOCK_SIZE = 256;
    dim3 block(BLOCK_SIZE);
    dim3 grid(params.batchSize);
    acceptDraftTokensByIdsWithPaths<T, BLOCK_SIZE><<<grid, block, 0, params.stream>>>(params.outputIds, params.draftIds,
        params.targetIds, params.sequenceLengths, params.acceptedLengths, params.finishedFinal, params.batchSlots,
        params.paths, params.endIds, params.medusaLogits, params.logitsPtrs, params.curTokensPerStep,
        params.targetTokensPerStep, params.bestPathIds, params.batchSize, params.vocabSize, params.maxBatchSize,
        params.maxSeqLen, params.maxDraftPathLen, params.maxDecodingTokens);
}

template void acceptDraftTokensByIdsWithPaths(AcceptDraftTokensByIdsWithPathsParams<float> const& params);
template void acceptDraftTokensByIdsWithPaths(AcceptDraftTokensByIdsWithPathsParams<__half> const& params);

} // namespace tensorrt_llm::kernels::speculative_decoding
